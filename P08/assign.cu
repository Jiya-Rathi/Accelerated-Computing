#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <assert.h>
#include <chrono>
#include <time.h>

#define PERTURB_B_VECTOR 1
#define PRINT_SOLUTION_SAMPLE 1

inline void checkCuda(hipError_t result, char const *const func, const int line)
{
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n",
                __FILE__, line, static_cast<unsigned int>(result),
                hipGetErrorName(result), func);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}
#define CHECK_CUDA(val) checkCuda((val), #val, __LINE__)

inline void checkCusolver(hipsolverStatus_t status, char const *const func, const int line)
{
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        fprintf(stderr, "cuSolver error at %s:%d code=%d \n",
                __FILE__, line, static_cast<int>(status));
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}
#define CHECK_CUSOLVER(val) checkCusolver((val), #val, __LINE__)

void buildHilbertMatrix(double* A, int N)
{
    for(int col = 0; col < N; col++) {
        for(int row = 0; row < N; row++) {
            A[row + col*N] = 1.0 / (double)(row + col + 1);
        }
    }
}

void buildBVector(double* B, int N)
{
    for(int i = 0; i < N; i++) {
        B[i] = 1.0;
    }
}

void perturbBVector(double* B, int N)
{
    for(int i = 0; i < N; i++) {
        double eps = (double)rand() / (double)RAND_MAX;
        B[i] += eps;
    }
}

int main(int argc, char* argv[])
{
    srand((unsigned int)time(NULL));

    hipsolverHandle_t cusolverH = nullptr;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));

    const int pivot_on = 0;

    int maxPower = 10;
    if (argc > 1) {
        maxPower = atoi(argv[1]);
    }

    printf("%-6s | %-25s | %-20s\n", "N", "Factor + Solve Time (ms)", "Solve Only Time (ms)");
    printf("--------------------------------------------------------------\n");

    for(int power = 1; power <= maxPower; power++) {
        int N = 1 << power;

        size_t sizeA = N * N * sizeof(double);
        size_t sizeB = N * sizeof(double);

        double* h_A = (double*) malloc(sizeA);
        double* h_B = (double*) malloc(sizeB);
        double* h_X = (double*) malloc(sizeB);

        buildHilbertMatrix(h_A, N);
        buildBVector(h_B, N);

        double *d_A = nullptr, *d_B = nullptr;
        int *d_info = nullptr;
        CHECK_CUDA(hipMalloc((void**)&d_A, sizeA));
        CHECK_CUDA(hipMalloc((void**)&d_B, sizeB));
        CHECK_CUDA(hipMalloc((void**)&d_info, sizeof(int)));

        CHECK_CUDA(hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice));

        int lwork = 0;
        CHECK_CUSOLVER(hipsolverDnDgetrf_bufferSize(cusolverH, N, N, d_A, N, &lwork));

        double* d_work = nullptr;
        CHECK_CUDA(hipMalloc((void**)&d_work, lwork * sizeof(double)));

        hipEvent_t start, stop;
        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));

        CHECK_CUDA(hipEventRecord(start));

        CHECK_CUSOLVER(hipsolverDnDgetrf(cusolverH, N, N, d_A, N, d_work, nullptr, d_info));
        CHECK_CUSOLVER(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, N, 1, d_A, N, nullptr, d_B, N, d_info));

        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop));

        float ms_factor_solve = 0.0f;
        CHECK_CUDA(hipEventElapsedTime(&ms_factor_solve, start, stop));
        CHECK_CUDA(hipMemcpy(h_X, d_B, sizeB, hipMemcpyDeviceToHost));

#if PRINT_SOLUTION_SAMPLE
        printf("N = %4d | x[0] = %.6f, x[N/2] = %.6f, x[N-1] = %.6f\n", N, h_X[0], h_X[N/2], h_X[N-1]);
#endif

#if PERTURB_B_VECTOR
        buildBVector(h_B, N);
        perturbBVector(h_B, N);
        CHECK_CUDA(hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice));
#endif

        CHECK_CUDA(hipEventRecord(start));
        CHECK_CUSOLVER(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, N, 1, d_A, N, nullptr, d_B, N, d_info));
        CHECK_CUDA(hipDeviceSynchronize());
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop));

        float ms_solve_only = 0.0f;
        CHECK_CUDA(hipEventElapsedTime(&ms_solve_only, start, stop));

        printf("%-6d | %-25f | %-20f\n", N, ms_factor_solve, ms_solve_only);

        CHECK_CUDA(hipFree(d_A));
        CHECK_CUDA(hipFree(d_B));
        CHECK_CUDA(hipFree(d_info));
        CHECK_CUDA(hipFree(d_work));
        CHECK_CUDA(hipEventDestroy(start));
        CHECK_CUDA(hipEventDestroy(stop));

        free(h_A);
        free(h_B);
        free(h_X);
    }

    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverH));
    hipDeviceReset();
    return 0;
}
