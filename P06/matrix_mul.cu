#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 32
#define FIXED_P 1024  // shared dimension

__global__ void matrixMultiply(const float* A, const float* B, float* C,
                               int m, int p, int pB, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < p; k++) {
            sum += A[row * p + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

__global__ void matrixTiledMultiply(const float* A, const float* B, float* C,
                                    int m, int p, int pB, int n)
{
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float val = 0.0f;
    int numTiles = (p + TILE_WIDTH - 1) / TILE_WIDTH;

    for (int t = 0; t < numTiles; t++) {
        int tiledACol = t * TILE_WIDTH + threadIdx.x;
        int tiledBRow = t * TILE_WIDTH + threadIdx.y;

        if (row < m && tiledACol < p)
            tileA[threadIdx.y][threadIdx.x] = A[row * p + tiledACol];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < n && tiledBRow < p)
            tileB[threadIdx.y][threadIdx.x] = B[tiledBRow * n + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            val += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < m && col < n) {
        C[row * n + col] = val;
    }
}

int main(int argc, char** argv)
{
    if (argc != 2) {
        printf("Usage: %s <m_size>\n", argv[0]);
        return 1;
    }

    int m = atoi(argv[1]);
    int p = FIXED_P;
    int n = m;

    printf("Matrix A: %dx%d, B: %dx%d, C: %dx%d\n", m, p, p, n, m, n);

    size_t sizeA = m * p * sizeof(float);
    size_t sizeB = p * n * sizeof(float);
    size_t sizeC = m * n * sizeof(float);

    float *hA = (float*)malloc(sizeA);
    float *hB = (float*)malloc(sizeB);
    float *hC = (float*)malloc(sizeC);
    float *hC_tiled = (float*)malloc(sizeC);

    if (!hA || !hB || !hC || !hC_tiled) {
        printf("Host memory allocation failed!\n");
        return 1;
    }

    srand(0);
    for (int i = 0; i < m * p; i++) hA[i] = (float)rand() / RAND_MAX;
    for (int i = 0; i < p * n; i++) hB[i] = (float)rand() / RAND_MAX;

    float *dA, *dB, *dC;
    if (hipMalloc(&dA, sizeA) != hipSuccess ||
        hipMalloc(&dB, sizeB) != hipSuccess ||
        hipMalloc(&dC, sizeC) != hipSuccess) {
        printf("Device memory allocation failed — skipping size m=%d\n", m);
        return 1;
    }

    hipMemcpy(dA, hA, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeB, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 block(TILE_WIDTH, TILE_WIDTH);
    dim3 grid((n + TILE_WIDTH - 1) / TILE_WIDTH,
              (m + TILE_WIDTH - 1) / TILE_WIDTH);

    // Naive
    hipMemset(dC, 0, sizeC);
    hipEventRecord(start);
    matrixMultiply<<<grid, block>>>(dA, dB, dC, m, p, p, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float naiveMs = 0.0f;
    hipEventElapsedTime(&naiveMs, start, stop);
    hipMemcpy(hC, dC, sizeC, hipMemcpyDeviceToHost);

    // Tiled
    hipMemset(dC, 0, sizeC);
    hipEventRecord(start);
    matrixTiledMultiply<<<grid, block>>>(dA, dB, dC, m, p, p, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tiledMs = 0.0f;
    hipEventElapsedTime(&tiledMs, start, stop);
    hipMemcpy(hC_tiled, dC, sizeC, hipMemcpyDeviceToHost);

    printf("  Naive kernel time: %.3f ms\n", naiveMs);
    printf("  Tiled kernel time: %.3f ms\n\n", tiledMs);

    hipFree(dA); hipFree(dB); hipFree(dC);
    hipEventDestroy(start); hipEventDestroy(stop);
    free(hA); free(hB); free(hC); free(hC_tiled);

    return 0;
}
